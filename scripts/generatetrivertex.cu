// reading a text file

#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>

void read_from_triangle(std::string name, int &pnumber, int &tnumber, double *&points, int *&triangles, int *&neigh){
    std::string line;
    std::ifstream nodefile(name + ".node");
    double a1, a2, a3, a4;
    int i = 0;
    
    //std::cout<<"Node file"<<std::endl;
    if (nodefile.is_open())
    {
        nodefile >> pnumber ;
        //std::cout<<pnumber<<std::endl;

        std::getline(nodefile, line); 
        points = (double *)malloc(2*pnumber*sizeof(double));
        while (nodefile >> a1 >> a2 >> a3 >> a4)
        {
            points[2*i + 0] = a2;
            points[2*i + 1] = a3;
            //std::cout<<points[2*i + 0]<<" "<<points[2*i + 1]<<std::endl;
            i++;
            //std::cout<<a2<<" "<<a3<<std::endl;
            
        }
        
    }
    else 
        std::cout << "Unable to open node file"; 

    nodefile.close();


    //std::cout<<"Ele file"<<std::endl;
    std::ifstream elefile(name + ".ele");
    int t1, t2, t3, t4;
    i = 0;
    if(elefile.is_open()){
        elefile >> tnumber ;
        triangles = (int *)malloc(3*tnumber*sizeof(int));
        std::getline(elefile, line); 
        while (elefile >> t1 >> t2 >> t3 >> t4 )
        {
            //std::cout<<t2<<" "<<t3<<" "<<t4<<std::endl;
            triangles[3*i + 0] = t2;
            triangles[3*i + 1] = t3;
            triangles[3*i + 2] = t4;
            //std::cout<<triangles[3*i + 0]<<" "<<triangles[3*i + 1]<<" "<<triangles[3*i + 2]<<std::endl;
            i++;
        }
    }else std::cout << "Unable to open ele file";

    elefile.close();

    //std::cout<<"Neigh file"<<std::endl;
    std::ifstream neighfile(name + ".neigh");
    i = 0;
    if(neighfile.is_open()){
        std::getline(neighfile, line); 
        neigh =(int *)malloc(3*tnumber*sizeof(int));
        while (neighfile >> t1 >> t2 >> t3 >> t4 )
        {
            neigh[3*i + 0] = t2;
            neigh[3*i + 1] = t3;
            neigh[3*i + 2] = t4;
            //std::cout<<t2<<" "<<t3<<" "<<t4<<std::endl;
            i++;
        }
    }else std::cout << "Unable to open neigh file";
    neighfile.close();
}

__global__ void initialize_memory(int* cu_trivertex, int* cu_triangles, int pnumber, int tnumber){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j;
    if(i < pnumber){  
        for (j = 0; j < tnumber; j++){
            if(i == cu_triangles[3*j + 0] ||  i == cu_triangles[3*j + 1] || i == cu_triangles[3*j + 2]){
                cu_trivertex[i] = j;
                break;
            }
        }
    }
}

int main(int argc, char* argv[]) {
    double *points;
    int *triangles;
    int *neigh;
    int *trivertex;
    int *cu_triangles;
    int *cu_trivertex;

    int pnumber, tnumber;
    std::string name(argv[1]);
	std::cout<<name<<std::endl;
	read_from_triangle(name, pnumber, tnumber, points, triangles, neigh);
	std::cout << " " << tnumber << " " << pnumber << "\n";

    trivertex = (int *)malloc(pnumber*sizeof(int));
	hipMalloc((void**) &cu_triangles, 3*tnumber*sizeof(int));
	hipMemcpy(cu_triangles, triangles, 3*tnumber*sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void**) &cu_trivertex, pnumber*sizeof(int));
	int numThreads = 128;  // max register per block is 65536, 65536/512
    std::cout<<"generating trivertex"<<std::endl;
	initialize_memory<<<(pnumber + (numThreads-1))/numThreads, numThreads>>>( cu_trivertex, cu_triangles, pnumber, tnumber);
    
	hipMemcpy(trivertex, cu_trivertex, pnumber*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(cu_trivertex);
	hipFree(cu_triangles);
    
    std::cout<<"storing .trivertex"<<std::endl;
    std::ofstream myfile;
    myfile.open (name + ".trivertex");
    myfile << pnumber<<"\n";
    for (size_t i = 0; i < pnumber; i++)
    {
        myfile << i<<" "<<trivertex[i]<<"\n";
    }
    
    myfile.close();

    return 0;
}
